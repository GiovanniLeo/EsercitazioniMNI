#include "hip/hip_runtime.h"
//#define VS
#define MGPU

#ifdef VS
#include "hip/hip_runtime.h"
#include ""

#endif

#ifdef MGPU
#include<hip/hip_runtime.h>
#endif

#include <stdio.h>
#include<stdlib.h>

//#define DEBUG_MATRIX
#define DEBUG


//Dal profiler nvida per il kernel vengono usati  registri (si deve passare sulla multiGPU)

void initializeArray(int *array, int N);
void printArray(int *array, int N);
__global__ void scalarProductv1GPU(int *a, int*b, int *c, int N);
int scalarProductCPU(int *a, int*b, int N);

int main()
{
	int *aHost, *bHost,*rHost;
	int *aDevice, *bDevice, *cDevice;
	int size, N, productCPU = 0,productGPU = 0, i;
	dim3 gridDim, blockDim;
	hipEvent_t start, stop;
	float elapsed, elapsedCPU,elapsedGPU;


	printf("Inserisci la size degli array(N):");
	fflush(stdout);
	scanf("%d", &N);

	printf("Inserisci la size dei blocchi di thread (Nt):");
	fflush(stdout);
	scanf("%d", &blockDim.x);

	//Determino il numero esatto di blocchi
	gridDim.x = N / blockDim.x + ((N % blockDim.x) == 0 ? 0 : 1);

	size = N * sizeof(int);

#ifdef DEBUG
	printf("Size della matrice: %d\n", N);
	printf("Numero totale di blocchi: %d\n", gridDim.x);
	printf("Numero totale dei Thread per blocco: %d\n", blockDim.x);
#endif

	//Alloco la memoria sull' host
	aHost = (int*)malloc(size);
	bHost = (int*)malloc(size);
	rHost = (int*)calloc(N, sizeof(int)); //Azzeriamo gli array che raccolgono il risultato(lo inizializzo a zero)

	//Alloco memoria sul Device
	hipMalloc((void **)&aDevice, size);
	hipMalloc((void **)&bDevice, size);
	hipMalloc((void **)&cDevice, size);

	//Azzeriamo gli array che raccolgono il risultato
	hipMemset(cDevice, 0, size);

	//inizializzo gli array
	initializeArray(aHost, N);
	initializeArray(bHost, N);

#ifdef DEBUG
	printArray(aHost, N);
	printArray(bHost, N);
#endif

	//copio i dati dall'host al device
	hipMemcpy(aDevice, aHost, size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, size, hipMemcpyHostToDevice);

	//Somma parallela
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	scalarProductv1GPU << <gridDim, blockDim >> > (aDevice, bDevice, cDevice, N);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedGPU, start, stop);

	//De-allocazione eventi
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(rHost, cDevice, size, hipMemcpyDeviceToHost);


	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for ( i = 0; i < N; i++)
	{
		productGPU += rHost[i];
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedCPU, start, stop);

	printf("Tempo per prodotto scalare GPU:%f ms\n", elapsedGPU + elapsedCPU);

	//Somma Seriale
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	productCPU = scalarProductCPU(aHost, bHost, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Tempo per prodotto scalare CPU:%f ms\n", elapsed);
	//fare il metodo di controllo controllo

	printf("Prodotto scalare CPU: %d\n",productCPU);
	printf("Prodotto scalare GPU: %d\n", productGPU);

	//De-allocazione eventi
	hipEventDestroy(start); hipEventDestroy(stop);
	//De-allocazione Host
	free(aHost); free(bHost); free(rHost);
	//De-allocazione Device
	hipFree(aDevice); hipFree(bDevice); hipFree(cDevice);

}


void initializeArray(int *array, int N)
{
	int i;
	for (i = 0; i < N; i++)
	{
		array[i] = rand() % 5;
	}

}

void printArray(int *array, int N)
{
	int i;
	if (N < 20)
	{
		for ( i = 0; i < N; i++)
		{
			printf("%d ", array[i]);
		}
		printf("\n");
	}
}

__global__ void scalarProductv1GPU(int *a, int*b, int *c, int N)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (id < N)
	{

		c[id] = a[id] * b[id];
	}
}

int scalarProductCPU(int *a, int*b, int N)
{
	int i,product = 0;

	for ( i = 0; i < N; i++)
	{
		product += a[i] * b[i];
	}

	return product;
}
