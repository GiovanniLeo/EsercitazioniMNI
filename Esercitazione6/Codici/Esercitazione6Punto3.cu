#include "hip/hip_runtime.h"
#define VS
//#define MGPU

#ifdef VS
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#endif

#ifdef MGPU
#include<hip/hip_runtime.h>
#endif

#include <stdio.h>
#include<stdlib.h>

//#define DEBUG_MATRIX
#define DEBUG


//Dal profiler nvida per il kernel vengono usati  registri (si deve passare sulla multiGPU)Ese

void initializeArray(int *array, int N);
void printArray(int *array, int N);
__global__ void scalarProductv2GPU(int *a, int*b, int *c, int N);
int scalarProductCPU(int *a, int*b, int N);

int main()
{
	int *aHost, *bHost, *rHost;
	int *aDevice, *bDevice, *cDevice;
	int size, sizeSM, N, productCPU = 0, productGPU = 0, i;
	dim3 gridDim, blockDim;
	hipEvent_t start, stop;
	float elapsed;


	printf("Inserisci la size degli array(N):");
	fflush(stdout);
	scanf("%d", &N);

	printf("Inserisci la size dei blocchi di thread (Nt):");
	fflush(stdout);
	scanf("%d", &blockDim.x);

	//Determino il numero esatto di blocchi 
	gridDim.x = N / blockDim.x + ((N % blockDim.x) == 0 ? 0 : 1);

	size = N * sizeof(int);

#ifdef DEBUG
	printf("Size della matrice: %d\n", N);
	printf("Numero totale di blocchi: %d\n", gridDim.x);
	printf("Numero totale dei Thread per blocco: %d\n", blockDim.x);
#endif 

	//Alloco la memoria sull' host
	aHost = (int*)malloc(size);
	bHost = (int*)malloc(size);
	rHost = (int*)calloc(N, sizeof(int)); //Azzeriamo gli array che raccolgono il risultato(lo inizializzo a zero)

	//Alloco memoria sul Device
	hipMalloc((void **)&aDevice, size);
	hipMalloc((void **)&bDevice, size);
	hipMalloc((void **)&cDevice, gridDim.x * sizeof(int)); //C deve avere un numero di elemnti pari al numero di blocchi

	//Azzeriamo gli array che raccolgono il risultato
	hipMemset(cDevice, 0, size);

	//inizializzo gli array
	initializeArray(aHost, N);
	initializeArray(bHost, N);

#ifdef DEBUG
	printArray(aHost, N);
	printArray(bHost, N);
#endif 

	//copio i dati dall'host al device
	hipMemcpy(aDevice, aHost, size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, size, hipMemcpyHostToDevice);

	//Somma parallela
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	sizeSM = blockDim.x * sizeof(int);
	scalarProductv2GPU << <gridDim, blockDim, sizeSM >> > (aDevice, bDevice, cDevice, N);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(rHost, cDevice, gridDim.x * sizeof(int), hipMemcpyDeviceToHost);

	for (i = 0; i < gridDim.x; i++)
	{
		productGPU += rHost[i];
	}


	hipEventElapsedTime(&elapsed, start, stop);
	//De-allocazione eventi
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Tempo per prodotto scalare GPU:%f ms\n", elapsed);

	//Somma Seriale
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	productCPU = scalarProductCPU(aHost, bHost, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Tempo per prodotto scalare CPU:%f ms\n", elapsed);
	//fare il metodo di controllo controllo 

	printf("Prodotto scalare CPU: %d\n", productCPU);
	printf("Prodotto scalare GPU: %d\n", productGPU);

	//De-allocazione eventi
	hipEventDestroy(start); hipEventDestroy(stop);
	//De-allocazione Host
	free(aHost); free(bHost); free(rHost);
	//De-allocazione Device
	hipFree(aDevice); hipFree(bDevice); hipFree(cDevice);

}


void initializeArray(int *array, int N)
{
	int i;
	for (i = 0; i < N; i++)
	{
		array[i] = rand() % 5;
	}

}

void printArray(int *array, int N)
{
	int i;
	if (N < 20)
	{
		for (i = 0; i < N; i++)
		{
			printf("%d ", array[i]);
		}
		printf("\n");
	}
}

__global__ void scalarProductv2GPU(int *a, int*b, int *c, int N)
{

	extern __shared__ int s[];
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < N)
		s[threadIdx.x] = a[index] * b[index];

	__syncthreads();
	for (unsigned int dist = blockDim.x/2; dist > 0; dist >>= 1) {
		if (threadIdx.x < dist) {
			s[threadIdx.x] += s[threadIdx.x + dist];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		c[blockIdx.x] = s[threadIdx.x];
	}

}

int scalarProductCPU(int *a, int*b, int N)
{
	int i, product = 0;

	for (i = 0; i < N; i++)
	{
		product += a[i] * b[i];
	}

	return product;
}


