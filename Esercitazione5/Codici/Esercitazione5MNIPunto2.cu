#define VS
//#define MGPU

#ifdef VS
	#include "hip/hip_runtime.h"
	

#endif

#ifdef MGPU
	#include<cuda.h>
#endif

#include <stdio.h>
#include<stdlib.h>

//#define DEBUG_MATRIX
#define DEBUG


//Dal profiler nvida per il kernel vengono usati 13 registri (si deve passare sulla multiGPU)

void initializeMatrix(int *array, int matrixSize);
void printMatrix(int *matrix, int rows, int cols);
__global__ void sumMatrixGPU(int *a, int*b, int *c, int rows, int cols);
void sumMatrixCPU(int *a, int*b, int *c, int rows, int cols);
void matrixEqualCheck(int *mHost, int *mDevice, int cols, int rows);
int main()
{
	int *aHost, *bHost, *cHost, *rHost;
	int *aDevice, *bDevice, *cDevice;
	int size, N;
	dim3 gridDim, blockDim;
	int matrixSize;
	hipEvent_t start, stop;
	float elapsed;


	printf("Inserisci la size della matrice quadrata(N):");
	fflush(stdout);
	scanf("%d",&N);
	
	printf("Inserisci la size dei blocchi di thread Nt (NtxNt):");
	fflush(stdout);
	scanf("%d", &blockDim.x);

	blockDim.y = blockDim.x;
	
	matrixSize = N * N;

	//Determino il numero esatto di blocchi 
	gridDim.x = N / blockDim.x + ((N % blockDim.x) == 0 ? 0 : 1);
	gridDim.y = N / blockDim.y + ((N % blockDim.y) == 0 ? 0 : 1);

	size = matrixSize * sizeof(int);

#ifdef DEBUG
	printf("Size della matrice: %d\n", matrixSize);
	printf("Numero totale di blocchi: %d (%d,%d)\n",gridDim.x*gridDim.y, gridDim.x, gridDim.y);
	printf("Numero totale dei Thread per blocco: %d (%d,%d)\n",blockDim.x*blockDim.y, blockDim.x, blockDim.y);
#endif 

	//Alloco la memoria sull' host
	aHost = (int*)malloc(size);
	bHost = (int*)malloc(size);
	rHost = (int*)malloc(size);
	cHost = (int*)calloc(matrixSize, sizeof(int)); //Azzeriamo gli array che raccolgono il risultato(lo inizializzo a zero)

	//Alloco memoria sul Device
	hipMalloc((void **)&aDevice, size);
	hipMalloc((void **)&bDevice, size);
	hipMalloc((void **)&cDevice, size);

	//Azzeriamo gli array che raccolgono il risultato
	hipMemset(cDevice, 0, size);

	//inizializzo le matrici
	initializeMatrix(aHost, matrixSize);
	initializeMatrix(bHost, matrixSize);

	//copio i dati dall'host al device
	hipMemcpy(aDevice, aHost, size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, size, hipMemcpyHostToDevice);

	//Somma parallela
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	sumMatrixGPU <<<gridDim,blockDim>>> (aDevice, bDevice, cDevice, N, N);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsed, start, stop);
	//De-allocazione eventi
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	printf("Tempo per la somma di matrici GPU:%f ms\n", elapsed);

	//Somma Seriale
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	sumMatrixCPU(aHost, bHost, cHost, N, N);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("Tempo per la somma di matrici CPU:%f ms\n", elapsed);
	//fare il metodo di controllo controllo 
	hipMemcpy(rHost, cDevice, size, hipMemcpyDeviceToHost);
#ifdef DEBUG_MATRIX
	printf("Host\n");
	printMatrix(cHost, N, N);
	printf("Device\n");
	printMatrix(rHost, N, N);
#endif 
	
	matrixEqualCheck(cHost,rHost,N,N);

	//De-allocazione eventi
	hipEventDestroy(start); hipEventDestroy(stop);
	//De-allocazione Host
	free(aHost); free(bHost); free(cHost); free(rHost);
	//De-allocazione Device
	hipFree(aDevice); hipFree(bDevice); hipFree(cDevice);

}


void initializeMatrix(int *array, int matrixSize)
{
	int i;
	for ( i = 0; i < matrixSize; i++)
	{
		array[i] = i;
	}
	
}

__global__ void sumMatrixGPU(int *a, int*b, int *c, int rows, int cols)
{
	//definire id per il controllo
	int idx = (blockDim.x*blockIdx.x) + threadIdx.x;
	int idy = (blockDim.y*blockIdx.y) + threadIdx.y;
	if (idx < rows && idy < cols)
	{
		//Essendo che rows e cols sono uguli li possiamo usare tranquillamente in maniera intercambiabile
		c[(idx*cols) + idy] = a[(idx*cols) + idy] + b[(idx*cols) + idy];
	}
	
}

void printMatrix(int *matrix, int rows, int cols)
{
	int i, j;

	if (cols < 10 && rows < 10)
	{
		for (i = 0; i < rows; i++)
		{
			for (j = 0; j < cols; j++)
			{
				printf("%d ", matrix[(i*cols) + j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	printf("\n");

}

void sumMatrixCPU(int *a, int*b, int *c, int rows, int cols)
{
	int i, size;
	size = rows * cols;
	for (i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

void matrixEqualCheck(int *mHost, int *mDevice, int cols, int rows)
{
	int i, size, count = 0;
	size = cols * rows;

	for ( i = 0; i < size; i++)
	{
		if (mHost[i] == mDevice[i])
		{
			count++;
		}
	}

	if (count == size)
	{
		printf("Le matrici sono uguali\n");
	}
	else
	{
		printf("Le matrici sono diverse\n");
	}
}
