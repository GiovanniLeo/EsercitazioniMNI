#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<stdlib.h>
#include <stdio.h>
#include<hip/hip_runtime.h>

void initializeArray(int*, int);
void stampaArray(int*, int);
void equalArray(int*, int*, int);
void prodottoArrayCompPerCompCPU(int *, int *, int *, int);
__global__ void prodottoArrayCompPerCompGPU(int*, int*, int*, int);

int main(int argn, char * argv[])
{
	//numero di blocchi e numero di thread per blocco
	dim3 gridDim, blockDim;
	int N; //numero totale di elementi dell'array
	//array memorizzati sull'host
	int *A_host, *B_host, *C_host;
	//array memorizzati sul device
	int *A_device, *B_device, *C_device;
	int *copy; //array in cui copieremo i risultati di C_device
	int size; //size in byte di ciascun array

	printf("***\t PRODOTTO COMPONENTE PER COMPONENTE DI DUE ARRAY \t***\n");
	/* se l'utente non ha inserito un numero sufficiente di
	parametri da riga di comando, si ricorre ai valori di
	default per impostare il numero di thread per blocco, il
	numero totale di elementi e il flag di stampa */
	printf("***\t PRODOTTO COMPONENTE PER COMPONENTE DI DUE ARRAY \t***\n");
	printf("Inserisci il numero elementi dei vettori\n");
	scanf("%d", &N);
	printf("Inserisci il numero di thread per blocco\n");
	scanf("%d", &blockDim);


	//determinazione esatta del numero di blocchi
	gridDim = N / blockDim.x +
		((N%blockDim.x) == 0 ? 0 : 1);
	//size in byte di ogni array
	size = N * sizeof(int);

	//stampa delle info sull'esecuzione del kernel
	printf("Numero di elementi = %d\n", N);
	printf("Numero di thread per blocco = %d\n",
		blockDim.x);
	printf("Numero di blocchi = %d\n", gridDim.x);

	//allocazione dati sull'host
	A_host = (int*)malloc(size);
	B_host = (int*)malloc(size);
	//lo inizializzo a zero
	C_host = (int*) calloc(N, sizeof(int));
	copy = (int*)malloc(size);
	//allocazione dati sul device
	hipMalloc((void**)&A_device, size);
	hipMalloc((void**)&B_device, size);
	hipMalloc((void**)&C_device, size);

	//inizializzazione dati sull'host
	initializeArray(A_host, N);
	initializeArray(B_host, N);



	//copia dei dati dall'host al device
	hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice);
	hipMemcpy(B_device, B_host, size, hipMemcpyHostToDevice);

	//azzeriamo il contenuto della matrice C
	
	hipMemset(C_device, 0, size);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	//invocazione del kernel
	prodottoArrayCompPerCompGPU << <gridDim, blockDim >> >
		(A_device, B_device, C_device, N);
	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
	float elapsed;
	// tempo tra i due eventi in millisecondi
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copia dei risultati dal device all'host
	hipMemcpy(copy, C_device, size, hipMemcpyDeviceToHost);

	printf("tempo GPU=%f\n", elapsed);


	// calcolo su CPU
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	//chiamata alla funzione seriale per il prodotto di due array
	prodottoArrayCompPerCompCPU(A_host, B_host, C_host, N);
	hipEventRecord(stop);
	hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("tempo CPU=%f\n", elapsed);


	//stampa degli array e dei risultati
	if (N < 20)
	{
		printf("array A\n"); stampaArray(A_host, N);
		printf("array B\n"); stampaArray(B_host, N);
		printf("Risultati host\n"); stampaArray(C_host, N);
		printf("Risultati device\n"); stampaArray(copy, N);
	}

	//test di correttezza
	//equalArray(copy, C_host, N);



	//de-allocazione host
	free(A_host);
	free(B_host);
	free(C_host);
	free(copy);
	//de-allocazione device
	hipFree(A_device);
	hipFree(B_device);
	hipFree(C_device);
	return 0;
}

void initializeArray(int *array, int n)
{
	int i;
	for (i = 0; i < n; i++)
		array[i] = i;
}
void stampaArray(int* array, int n)
{
	int i;
	for (i = 0; i < n; i++)
		printf("%d ", array[i]);
	printf("\n");
}
void equalArray(int* a, int*b, int n)
{
	int i = 0;
	while (a[i] == b[i])
		i++;
	if (i < n)
		printf("I risultati dell'host e del device sono diversi\n");
	else
		printf("I risultati dell'host e del device coincidono\n");
}

//Seriale
void prodottoArrayCompPerCompCPU
(int *a, int *b, int *c, int n)
{
	int i;
	for (i = 0; i < n; i++)
		c[i] = a[i] * b[i];
}
//Parallelo
__global__ void prodottoArrayCompPerCompGPU
(int* a, int* b, int* c, int n)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < n)
		c[index] = a[index] * b[index];
}
