
#include "hip/hip_runtime.h"

#include<stdlib.h>
#include <stdio.h>

void inizializeArray(int *array, int dim);
void printArray(int *array, int dim);
__global__ void arrayProductGPU(int *a, int *b, int *c, int dim);


//#define DEBUG

int main(int argc, char* argv[])
{
	dim3 gridDim, blockDim; //Dimensione griglia e dimensione blocco
	int N; //Elementi array
	int *aHost, *bHost, *cHost; //ArrayCPU
	int *aDevice, *bDevice, *cDevice; //ArrayGPU
	int *temp;
	int size, sum = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Inserisci il numero di elenti del vettore:");
	fflush(stdout);
	scanf("%d", &N);
	
	blockDim.x = 32;
	gridDim.x = N / blockDim.x + ((N % blockDim.x) == 0 ? 0 : 1); //serve a determinare quanti blocchi c sono esattamente
	size = sizeof(int)*N;

#ifdef DEBUG
	printf("Numero di blocchi: %d\n", gridDim.x);
	printf("Numero di elementi array: %d\n", N);
	printf("Numero di thread per blocco: %d\n", blockDim.x);
#endif  

	aHost = (int*)malloc(size);
	bHost = (int*)malloc(size);
	cHost = (int*)calloc(N,sizeof(int));
	temp = (int*)malloc(size);

	hipMalloc((void**)&aDevice, size);
	hipMalloc((void**)&bDevice, size);
	hipMalloc((void**)&cDevice, size);


	inizializeArray(aHost, N);
	inizializeArray(bHost, N);
	hipMemset(cDevice, 0, size);

#ifdef DEBUG
	printArray(aHost, N);
#endif

	hipMemcpy(aDevice, aHost, size ,hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, size, hipMemcpyHostToDevice);
	
	hipEventRecord(start,0);
	arrayProductGPU <<<gridDim, blockDim>>> (aDevice, bDevice, cDevice, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsed;
	hipEventElapsedTime(&elapsed,start, stop);
	printf("Il prototto � stato eseguito in :%f ms\n", elapsed);

	hipMemcpy(cHost, cDevice, size, hipMemcpyDeviceToHost);

#ifdef DEBUG
	printArray(cHost, N);
#endif

	int i;
	for (i = 0; i < N; i++)
	{
		sum += cHost[i];
	}
	printf("La somma � :%d\n", sum);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(cDevice);
	free(aHost);
	free(bHost);
	free(cHost);







 
    return 0;
}

void inizializeArray(int *array, int dim)
{
	int i;
	for (i = 0; i < dim; i++)
	{
		array[i] = i;
	}
}

void printArray(int *array, int dim)
{
	int i;
	for (i = 0; i < dim; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}

/*
	*a: primo elemento della moltiplicazione
	*b: secondo elemento della moltiplicazione
	*c: array contenete il risultato della moltiplicazione
	dim: dimensione degli array
*/

__global__ void arrayProductGPU(int *a, int *b, int *c, int dim)
{
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (index < dim) //Facciamo questo controllo poich� potrebbero essereci thre thread che non hanno associato nessun indice dell'array
	{
		c[index] = a[index] * b[index];
	}
}

