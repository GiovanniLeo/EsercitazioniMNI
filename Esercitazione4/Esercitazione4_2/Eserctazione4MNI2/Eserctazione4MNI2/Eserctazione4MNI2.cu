
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define DEBUG

void initializeMatrix(int *matrix, int rows, int cols);
void printMatrix(int *matrix, int rows, int cols);
__global__ void matrixSumGpu(int *a, int *b, int *c, int rows, int cols);


int main()
{
	int rows, cols;
	int matrixDim;
	int *aHost, *bHost,*cHost;
	int *aDevice, *bDevice,*cDevice;
	int size;
	dim3 gridDim, blockDim;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	

	printf("Inserire le dimensioni della matrice NxN(rows cols):");
	fflush(stdout);
	scanf("%d %d", &rows, &cols);

	matrixDim = rows * cols;
	size = matrixDim * sizeof(int);

	blockDim.x = 4; blockDim.y = 8; //blocchi di 32 thread
	gridDim.x = cols / blockDim.x + ((cols % blockDim.x) == 0 ? 0 : 1);
	gridDim.y = rows / blockDim.y + ((rows % blockDim.y) == 0 ? 0 : 1);

#ifdef DEBUG
	printf("Numero di blocchi lungo asse x: %d Numero di blocchi lungo asse y:%d\n", gridDim.x, gridDim.y);
	printf("Numero di elementi della matrice : %d\n", matrixDim);
	printf("Numero di thread lungo asse x: %d Numero di thread lungo asse y:%d\n", blockDim.x,blockDim.y);
#endif  

	aHost = (int*)malloc(size);
	bHost = (int*)malloc(size);
	cHost = (int*)calloc(matrixDim, sizeof(int));


	hipMalloc((void**)&aDevice, size);
	hipMalloc((void**)&bDevice, size);
	hipMalloc((void**)&cDevice, size);

	initializeMatrix(aHost, rows, cols);
	initializeMatrix(bHost, rows, cols);
	hipMemset(cDevice, 0, size);

#ifdef DEBUG
	printMatrix(aHost, rows, cols);
	printMatrix(bHost, rows, cols);
#endif
	hipMemcpy(aDevice, aHost, size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, bHost, size, hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	matrixSumGpu <<<gridDim, blockDim >>> (aDevice, bDevice, cDevice, rows, cols);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	float elasped;
	hipEventElapsedTime(&elasped, start, stop);
	printf("Tempo per la somma di matrici :%f ms\n",elasped);

	hipMemcpy(cHost, cDevice, size, hipMemcpyDeviceToHost);

#ifdef DEBUG
	printMatrix(cHost, rows, cols);
#endif

   

    return 0;
}

void initializeMatrix(int *matrix, int rows, int cols)
{
	int i, j;

		for (i = 0; i < rows; i++)
		{
			for (j = 0; j < cols; j++)
			{
				matrix[(i*cols) + j] = (i*rows)+j;
			}
		}
}

void printMatrix(int *matrix, int rows, int cols)
{
	int i, j;

	if (cols < 10 && rows < 10)
	{
		for (i = 0; i < rows; i++)
		{
			for (j = 0; j < cols; j++)
			{
				printf("%d ", matrix[(i*cols) + j]);
			}
			printf("\n");
		}
		printf("\n");
	}

}

/*
	*a:   primo elemento della somma
	*b:   secondo elemento della somma
	*c:	  array contenete il risultato della somma
	cols: numero della colonne
	rows: numero delle righe
*/
__global__ void matrixSumGpu(int *a, int *b, int *c, int rows, int cols)
{
	int i, j;
	//definire id per il controllo
	int idx = (blockDim.x*blockIdx.x) + threadIdx.x;
	int idy = (blockDim.y*blockIdx.y) + threadIdx.y;

	for (i = 0; i < rows; i++)
	{
		for (j = 0; j < cols; j++)
		{
			if (idx < cols && idy < rows)
			{
				c[(i*cols) + j] = a[(i*cols) + j] + b[(i*cols) + j];
			}
			
		}
	}
}
